#include "../common/book.h"
#include "../common/cpu_bitmap.h"
#include <iostream>

#define DIM 1000

struct hipComplex
{
    float r;
    float i;
    hipComplex(float a, float b) : r(a), i(b) {}
    float magnitude2(void) { return r * r + i * i; }
    hipComplex operator*(const hipComplex &a)
    {
        return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
    }
    hipComplex operator+(const hipComplex &a)
    {
        return hipComplex(r + a.r, i + a.i);
    }
};

int julia(int x, int y)
{
    const float scale = 1.5;
    float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
    float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);
    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);
    int i = 0;
    for (i = 0; i < 200; i++)
    {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }
    return 1;
}

void kernel(unsigned char *ptr)
{
    for (int y = 0; y < DIM; y++)
    {
        for (int x = 0; x < DIM; x++)
        {
            int offset = x + y * DIM;
            int juliaValue = julia(x, y);
            ptr[offset * 4 + 0] = 255 * juliaValue;
            ptr[offset * 4 + 1] = 0;
            ptr[offset * 4 + 2] = 0;
            ptr[offset * 4 + 3] = 255;
        }
    }
}

void output_image(unsigned char *ptr)
{
    std::cout << "P3\n"
                  << DIM << ' ' << DIM << "\n255\n";

    for (int y = 0; y < DIM; y++)
    {
        for (int x = 0; x < DIM; x++)
        {
            int offset = x + y * DIM;

            auto r = ptr[offset * 4 + 0];
            auto g = ptr[offset * 4 + 1];
            auto b = ptr[offset * 4 + 2];

            // Write the translated [0,255] value of each color component.
            std::cout << static_cast<int>(r) << ' '
                      << static_cast<int>(g) << ' '
                      << static_cast<int>(b) << '\n';
        }
    }
}

int main(void)
{
    CPUBitmap bitmap(DIM, DIM);
    unsigned char *ptr = bitmap.get_ptr();
    kernel(ptr);
    // I don't suppose execute display_and_exit() on wsl2, it needs many steps to configure environment
    // so I output the image to a ppm as what we have learned form raytracing lab
    // bitmap.display_and_exit();
    output_image(ptr);
}