#include "hip/hip_runtime.h"
#include "../common/book.h"

#define SIZE (100 * 1024 * 1024)
int main(void)
{

    unsigned char *buffer = (unsigned char *)big_random_block(SIZE);
    unsigned int histo[256];
    auto start_time = clock();
    for (int i = 0; i < 256; i++)
        histo[i] = 0;
    for (int i = 0; i < SIZE; i++)
        histo[buffer[i]]++;
    long histoCount = 0;
    for (int i = 0; i < 256; i++)
    {
        histoCount += histo[i];
    }

    auto end_time = clock();
    double cpu_time_used = ((double)(end_time - start_time)) / CLOCKS_PER_SEC;
    printf("time cost %f s\n", cpu_time_used);

    printf("Histogram Sum: %ld\n", histoCount);
    free(buffer);
    return 0;
}