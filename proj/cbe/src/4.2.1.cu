#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/timer.h"
#define N 50000

void add(int *a, int *b, int *c)
{
    int tid = 0; // this is CPU zero, so we start at zero
    while (tid < N)
    {
        c[tid] = a[tid] + b[tid];
        tid += 1; // we have one CPU, so we increment by one
    }
}

__global__ void gpuadd(int *a, int *b, int *c)
{
    int tid = blockIdx.x; // handle the data at this index
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int main(void)
{
    int a[N], b[N], c[N];
    // fill the arrays 'a' and 'b' on the CPU
    for (int i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }
    PriciseTimer timer;
    timer.start();
    add(a, b, c);
    timer.end();
    timer.print_time();
    // display the results
    // for (int i = 0; i < N; i++)
    // {
    //     printf("%d + %d = %d\n", a[i], b[i], c[i]);
    // }

    int *dev_a, *dev_b, *dev_c;
    // allocate the memory on the GPU
    HANDLE_ERROR(hipMalloc((void **)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **)&dev_c, N * sizeof(int)));
    // fill the arrays 'a' and 'b' on the CPU
    for (int i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int),
                            hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int),
                            hipMemcpyHostToDevice));
    timer.start();
    gpuadd<<<N, 1>>>(dev_a, dev_b, dev_c);
    timer.end();
    timer.print_time();
    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int),
                            hipMemcpyDeviceToHost));

    // // display the results
    // for (int i = 0; i < N; i++)
    // {
    //     printf("%d + %d = %d\n", a[i], b[i], c[i]);
    // }

    // free the memory allocated on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}